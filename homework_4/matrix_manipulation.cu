#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
// #include <stdlib.h>
#include <stdio.h>

#include "matrix_manipulation.cuh"

namespace {
constexpr int kThreadsPerBlock = 256;

// Credit: lecture notes
inline hipError_t checkCudaErr(hipError_t err, const char* msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg,
            hipGetErrorString(err));
  }
  return err;
}

__global__ void ThresholdKernel(const double* array, int data_size,
                                double threshold, double* result) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= data_size) {
    return;
  }

  const auto value = array[idx];

  result[idx] = value <= threshold ? 0 : 1;
}
}  // namespace

namespace homework_4 {
void matrix_multiply(const double* lhs, const double* rhs, int width,
                     int height, double* result) {}

void threshold(const double* array, int data_size, double threshold,
               double* result) {
  double* array_gpu;
  double* result_gpu;
  const int data_size_bytes = data_size * sizeof(double);
  hipMalloc(&array_gpu, data_size_bytes);
  hipMalloc(&result_gpu, data_size_bytes);
  hipMemcpy(array_gpu, array, data_size_bytes, hipMemcpyHostToDevice);
  checkCudaErr(hipGetLastError(), "hipMemcpy");

  const dim3 Threads(kThreadsPerBlock);
  const int block_x = (data_size / kThreadsPerBlock) + 1;
  const dim3 Blocks(block_x);
  ThresholdKernel<<<Blocks, Threads>>>(array_gpu, data_size, threshold,
                                       result_gpu);

  hipMemcpy(result, result_gpu, data_size_bytes, hipMemcpyDeviceToHost);
  checkCudaErr(hipGetLastError(), "hipMemcpy");

  hipFree(array_gpu);
  hipFree(result_gpu);
  checkCudaErr(hipGetLastError(), "hipFree");
}

void reversed_threshold(const double* array, int data_size, double threshold,
                        double* result) {}
void element_wise_sum(const double* lhs, const double* rhs, int data_size,
                      double* result) {}

double sum(double* array, int data_size) {
  double result = 0.0;
  return result;
}
}  // namespace homework_4