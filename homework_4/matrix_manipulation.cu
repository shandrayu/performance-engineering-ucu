#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
// #include <stdlib.h>
#include <stdio.h>

#include "matrix_manipulation.cuh"

namespace {
constexpr int kThreadsPerBlock = 256;

// Credit: lecture notes
inline hipError_t checkCudaErr(hipError_t err, const char* msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg,
            hipGetErrorString(err));
  }
  return err;
}

__global__ void ThresholdKernel(const double* array, std::size_t data_size,
                                double threshold, double* result) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= data_size) {
    return;
  }

  const auto value = array[idx];

  result[idx] = value < threshold ? 0 : 1;
}

__global__ void ReversedThresholdKernel(const double* array,
                                        std::size_t data_size, double threshold,
                                        double* result) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= data_size) {
    return;
  }

  const auto value = array[idx];

  result[idx] = value < threshold ? 1 : 0;
}

__global__ void ElementWiseSumKernel(const double* first_array,
                                     const double* second_array,
                                     std::size_t data_size, double* result) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= data_size) {
    return;
  }

  result[idx] = first_array[idx] + second_array[idx];
}
}  // namespace

namespace homework_4 {
template <typename T>
struct CudaArrayContainer {
  CudaArrayContainer(const T* data, std::size_t num_elements)
      : num_elements_(num_elements) {
    data_size_bytes_ = num_elements_ * sizeof(T);
    hipMalloc(&gpu_data_, data_size_bytes_);
    checkCudaErr(hipGetLastError(), "hipMalloc");
  }
  ~CudaArrayContainer() {
    hipFree(gpu_data_);
    checkCudaErr(hipGetLastError(), "hipFree");
  }
  T* GetGpuPtr() const { return gpu_data_; }
  std::size_t GetNumElements() const { return num_elements_; }
  std::size_t GetDataSizeBytes() const { return data_size_bytes_; }

  void CopyToHost(T* host_pointer) const {
    hipMemcpy(host_pointer, GetGpuPtr(), GetDataSizeBytes(),
               hipMemcpyDeviceToHost);
    checkCudaErr(hipGetLastError(), "hipMemcpy");
  }

  void CopyFromHost(const T* host_pointer) const {
    hipMemcpy(GetGpuPtr(), host_pointer, GetDataSizeBytes(),
               hipMemcpyHostToDevice);
    checkCudaErr(hipGetLastError(), "hipMemcpy");
  }

 private:
  T* gpu_data_;
  std::size_t num_elements_;
  std::size_t data_size_bytes_;
};

void matrix_multiply(const double* lhs, const double* rhs, int width,
                     int height, double* result) {}

void threshold(const double* array, std::size_t data_size, double threshold,
               double* result) {
  CudaArrayContainer<double> array_gpu(array, data_size);
  CudaArrayContainer<double> result_gpu(result, data_size);

  array_gpu.CopyFromHost(array);

  const dim3 Threads(kThreadsPerBlock);
  const int block_x = (data_size / kThreadsPerBlock) + 1;
  const dim3 Blocks(block_x);
  ThresholdKernel<<<Blocks, Threads>>>(array_gpu.GetGpuPtr(),
                                       array_gpu.GetNumElements(), threshold,
                                       result_gpu.GetGpuPtr());

  result_gpu.CopyToHost(result);
}

void reversed_threshold(const double* array, std::size_t data_size,
                        double threshold, double* result) {
  CudaArrayContainer<double> array_gpu(array, data_size);
  CudaArrayContainer<double> result_gpu(result, data_size);

  array_gpu.CopyFromHost(array);

  const dim3 Threads(kThreadsPerBlock);
  const int block_x = (data_size / kThreadsPerBlock) + 1;
  const dim3 Blocks(block_x);
  ReversedThresholdKernel<<<Blocks, Threads>>>(
      array_gpu.GetGpuPtr(), array_gpu.GetNumElements(), threshold,
      result_gpu.GetGpuPtr());

  result_gpu.CopyToHost(result);
}

void element_wise_sum(const double* first_array, const double* second_array,
                      std::size_t data_size, double* result) {
  CudaArrayContainer<double> first_array_gpu(first_array, data_size);
  CudaArrayContainer<double> second_array_gpu(second_array, data_size);
  CudaArrayContainer<double> result_gpu(result, data_size);

  first_array_gpu.CopyFromHost(first_array);
  second_array_gpu.CopyFromHost(second_array);

  const dim3 Threads(kThreadsPerBlock);
  const int block_x = (data_size / kThreadsPerBlock) + 1;
  const dim3 Blocks(block_x);
  ElementWiseSumKernel<<<Blocks, Threads>>>(
      first_array_gpu.GetGpuPtr(), second_array_gpu.GetGpuPtr(),
      first_array_gpu.GetNumElements(), result_gpu.GetGpuPtr());

  result_gpu.CopyToHost(result);
}

double sum(double* array, std::size_t data_size) {
  double result = 0.0;
  return result;
}
}  // namespace homework_4